
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <hipblas.h>

#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>


#include <chrono>

#define BLOCK_SIZE 256
#define MAX 1000000

/* this GPU kernel function is used to initialize the random states */
__global__ void init(unsigned int seed, hiprandState_t* states)
{
    int tid = (blockDim.x * blockIdx.x) + threadIdx.x;
    /* we have to initialize the state */
    hiprand_init(seed, /* the seed can be the same for each core, here we pass the time in from the CPU */
            tid, /* the sequence number should be different for each core (unless you want all
                    cores to get the same sequence of numbers for some reason - use thread id! */
            0, /* the offset is how much extra we advance in the sequence for each call, can be 0 */
            &states[tid]);
}

__global__ void randoms(hiprandState_t* states, float * numbers)
{
    int tid = (blockDim.x * blockIdx.x) + threadIdx.x;
    numbers[tid] = hiprand_uniform(&states[tid]) * MAX;
}

void genRandomNumbers(size_t numBlocks, int dataSetSize, float *randomArray)
{
  hiprandState_t* states;
  hipMalloc((void**) &states, dataSetSize * sizeof(hiprandState_t));

  init<<<numBlocks, BLOCK_SIZE>>>(time(0), states);
  randoms<<<numBlocks, BLOCK_SIZE>>>(states, randomArray);

  hipFree(states);
}

void runCublasExample(int dataSetSize)
{
    int numBlocks = dataSetSize / BLOCK_SIZE;

    float * randomArray;
    hipMalloc((void**) &randomArray, dataSetSize * sizeof(float));
    genRandomNumbers(numBlocks, dataSetSize, randomArray);

    hipblasStatus_t status;

    auto start = std::chrono::high_resolution_clock::now();
    float sum = cublasSasum(dataSetSize, randomArray, 1);
    auto stop = std::chrono::high_resolution_clock::now();

    status = cublasGetError();
    if (status != HIPBLAS_STATUS_SUCCESS) {
      fprintf (stderr, "!!!! kernel execution error.\n");
      return;
    }

    auto runTime = std::chrono::duration_cast<std::chrono::microseconds>(stop - start).count();
    printf("The random data set of size %d has an absolute value sum of %f. It took %dus to calculate\n", dataSetSize, sum, runTime);
    hipFree(randomArray);
}

// Main function
int main(int argc, char* argv[])
{
 
  runCublasExample(32768);
  runCublasExample(32768);
  runCublasExample(65536);
  runCublasExample(131072);
  return 0;
}
