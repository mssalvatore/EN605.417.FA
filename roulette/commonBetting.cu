
#include <hip/hip_runtime.h>
typedef int (*betsize_calculator_function)(int bettingFactor, int lossCount);
typedef int (*loss_calculator_function)(int currentLossCount, int spinResult, int winLossFactor[]);

__device__ void executeBettingStrategy(int * outPurse, int * outMaxPurse, int * outMinPurse, int64_t * outIntegral, loss_calculator_function calcLossCount, betsize_calculator_function calcBetSize, float winProbability, float* spinData, int spinsPerRun, int bettingFactor = 2, int startingBet = 1)
{
    int tid = (blockDim.x * blockIdx.x) + threadIdx.x;
    int row = tid * spinsPerRun;
    int purse = 0;
    int maxPurse = 0;
    int minPurse = 0;
    int64_t integral = 0;
    int betSize = startingBet;
    int lossCount = 0;
    int winLossFactor[] = {1, -1};
    int totalLosses = 0;

    for (int i = 0; i < spinsPerRun; i++)
    {
        //printf("!!Begin!! TID: %d -- Run: %d -- Purse: %d -- Bet: %d -- Losses: %d -- Spin: %f\n", tid, i, purse, betSize, lossCount, spinData[row+i]);
        int lostSpin = (spinData[row + i] >= winProbability);
        purse += winLossFactor[lostSpin] * betSize;
        integral += purse;

        maxPurse = ((purse > maxPurse) * purse) + ((purse < maxPurse) * maxPurse);
        minPurse = ((purse < minPurse) * purse) + ((purse > minPurse) * minPurse);

        lossCount = calcLossCount(lossCount, lostSpin, winLossFactor);
        totalLosses += lostSpin;
        betSize = calcBetSize(bettingFactor, lossCount);
        //printf("!!END  !! TID: %d -- Run: %d -- Purse: %d -- Bet: %d -- Losses: %d -- Spin: %f\n\n", tid, i, purse, betSize, lossCount, spinData[row+i]);
    }

    outPurse[tid] = purse;
    outMaxPurse[tid] = maxPurse;
    outMinPurse[tid] = minPurse;
    outIntegral[tid] = integral;


}

__device__ int lossCountResetOnWin(int currentLossCount, int spinResult, int * /*[] winLossFactor */)
{
    return currentLossCount * spinResult + spinResult;
}

