
#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <fstream>
#include <vector>
#include <cstring>
#include <iostream>

#define NUM_ROULETTE_SLOTS 37

__device__ __constant__ float cudaColorTranslation[NUM_ROULETTE_SLOTS];
float hostColorTranslation[NUM_ROULETTE_SLOTS] = {
    1,
    1,
    0,
    1,
    0,
    1,
    0,
    1,
    0,
    1,
    0,
    0,
    1,
    0,
    1,
    0,
    1,
    0,
    1,
    1,
    0,
    1,
    0,
    1,
    0,
    1,
    0,
    1,
    0,
    0,
    1,
    0,
    1,
    0,
    1,
    0,
    1
};

/* this GPU kernel function is used to initialize the random states */
__global__ void initRandom(unsigned int seed, hiprandState_t* states)
{
    int tid = (blockDim.x * blockIdx.x) + threadIdx.x;
    /* we have to initialize the state */
    hiprand_init(seed, /* the seed can be the same for each core, here we pass the time in from the CPU */
            tid, /* the sequence number should be different for each core (unless you want all
                    cores to get the same sequence of numbers for some reason - use thread id! */
            0, /* the offset is how much extra we advance in the sequence for each call, can be 0 */
            &states[tid]);
}

__global__ void genRandoms(hiprandState_t* states, float* numbers, int count)
{
    int tid = (blockDim.x * blockIdx.x) + threadIdx.x;
    int row = tid * count;
    for (int i = 0; i<count; i++)
    {
        numbers[row + i] = hiprand_uniform(&states[tid]);
    }
}

__device__ int integerPow(int num, int exponent)
{
    int result = 1;

    for (int i = 0; i < exponent; i++)
    {
        result *= num;
    }

    return result;
}

__host__ std::vector<int> readLineFromFile(std::ifstream * file)
{
    std::vector<int> nums;

    std::string line;
    if (! std::getline(*file, line)) {
        std::cout<<"CRAP\n";
        return nums;
    }

    char * dup = strdup(line.c_str());
    char delim[] = " ";
    char * token = std::strtok(dup, delim);
    while (token != NULL) {
        nums.push_back(atoi(token));
        token = std::strtok(NULL, delim);
    }


    return nums;
}

__global__ void translateSpinsToColors(float* spinData)
{
    int tid = (blockDim.x * blockIdx.x) + threadIdx.x;
    spinData[tid] = cudaColorTranslation[(int)spinData[tid]];
}

__host__ void getDimensions(int *numBlocks, int *numThreads, int maxThreadsPerBlock, int dataSize)
{
    *numThreads = maxThreadsPerBlock;
    while (((dataSize % *numThreads) != 0) && (*numThreads > 0)) {
        (*numThreads)--;
    }

    if (*numThreads == 0) {
        *numBlocks = 0;
        *numThreads = 0;
    }

    *numBlocks = dataSize / *numThreads;
}

