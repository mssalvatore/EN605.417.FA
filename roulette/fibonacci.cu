#include "hip/hip_runtime.h"
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

__constant__  static const float PHI = 1.618033;

// http://www.maths.surrey.ac.uk/hosted-sites/R.Knott/Fibonacci/fibFormula.html
__device__ int calculateFibonacciBetSize(int bettingFactor, int lossCount)
{
    int n = lossCount + 1;
    return bettingFactor * (round((pow(PHI, n) - pow(-PHI, -n)) / sqrtf(5))) ;
}

__global__ void fibonacci(int * outPurse, int * outMaxPurse, int * outMinPurse, int64_t * outIntegral, float winProbability, float* spinData, int spinsPerRun, int bettingFactor = 1)
{
    executeBettingStrategy(outPurse, outMaxPurse, outMinPurse, outIntegral, &lossCountResetOnWin, &calculateFibonacciBetSize, winProbability, spinData, spinsPerRun, bettingFactor);
}

