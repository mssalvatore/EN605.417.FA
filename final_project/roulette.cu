#include "hip/hip_runtime.h"
#include <chrono>
#include <cmath>
#include <cstdlib>
#include <exception>
#include <iostream>
#include <stdlib.h>
#include <stdio.h>
#include <sstream>
#include <time.h>

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include "options.cpp"

#define MAX_THREADS_PER_BLOCK 1024
#define BLOCK_SIZE MAX_THREADS_PER_BLOCK
#define WARP_SIZE 32

__constant__  static const float PHI = 1.618033;

/* this GPU kernel function is used to initialize the random states */
__global__ void initRandom(unsigned int seed, hiprandState_t* states)
{
    int tid = (blockDim.x * blockIdx.x) + threadIdx.x;
    /* we have to initialize the state */
    hiprand_init(seed, /* the seed can be the same for each core, here we pass the time in from the CPU */
            tid, /* the sequence number should be different for each core (unless you want all
                    cores to get the same sequence of numbers for some reason - use thread id! */
            0, /* the offset is how much extra we advance in the sequence for each call, can be 0 */
            &states[tid]);
}

// Generate numbers randomly
__device__ void genRandoms(hiprandState_t* states, float* numbers, int count)
{
    int tid = (blockDim.x * blockIdx.x) + threadIdx.x;
    int row = tid * count;
    for (int i = 0; i<count; i++)
    {
        numbers[row + i] = hiprand_uniform(&states[tid]);
    }
}

__device__ int integerPow(int num, int exponent)
{
    int result = 1;

    for (int i = 0; i < exponent; i++)
    {
        result *= num;
    }

    return result;
}

__global__ void martingale(float winProbability, hiprandState_t* states, float* spinData, int spinsPerRun, int bettingFactor = 2)
{
    genRandoms(states, spinData, spinsPerRun);

    int tid = (blockDim.x * blockIdx.x) + threadIdx.x;
    int row = tid * spinsPerRun;
    int purse = 0;
    int betSize = 1;
    int lossCount = 0;
    int winLossFactor[] = {1, -1};
    int totalLosses = 0;

    printf("Win probability: %f\n\n", winProbability);
    for (int i = 0; i < spinsPerRun; i++)
    {
        printf("!!Begin!! TID: %d -- Run: %d -- Purse: %d -- Bet: %d -- Losses: %d -- Spin: %f\n", tid, i, purse, betSize, lossCount, spinData[row+i]);
        int lostSpin = (spinData[row + i] >= winProbability);
        purse += winLossFactor[lostSpin] * betSize;

        lossCount = lossCount * lostSpin + lostSpin;
        totalLosses += (lossCount > 0);
        betSize = integerPow(bettingFactor, lossCount);
        printf("!!END  !! TID: %d -- Run: %d -- Purse: %d -- Bet: %d -- Losses: %d -- Spin: %f\n\n", tid, i, purse, betSize, lossCount, spinData[row+i]);
    }

    printf("Purse: %d\n", purse);
    printf("TotalLosses %d\n", totalLosses);
}

__global__ void dalembert(float winProbability, hiprandState_t* states, float* spinData, int spinsPerRun, int bettingFactor = 1)
{
    genRandoms(states, spinData, spinsPerRun);

    int tid = (blockDim.x * blockIdx.x) + threadIdx.x;
    int row = tid * spinsPerRun;
    int purse = 0;
    int betSize = bettingFactor;
    int lossCount = 0;
    int winLossFactor[] = {1, -1};

    printf("Win probability: %f\n\n", winProbability);
    for (int i = 0; i < spinsPerRun; i++)
    {
        printf("!!Begin!! TID: %d -- Run: %d -- Purse: %d -- Bet: %d -- Losses: %d -- Spin: %f\n", tid, i, purse, betSize, lossCount, spinData[row+i]);
        int lostSpin = (spinData[row + i] >= winProbability);
        int wonSpin = !(spinData[row + i] >= winProbability);
        purse += winLossFactor[lostSpin] * betSize;

        //lossCount = lossCount * lostSpin + lostSpin;
        lossCount = (lossCount + winLossFactor[wonSpin]);
        lossCount *= (lossCount > 0);
        betSize = bettingFactor + (bettingFactor * lossCount);
        printf("!!END  !! TID: %d -- Run: %d -- Purse: %d -- Bet: %d -- Losses: %d -- Spin: %f\n\n", tid, i, purse, betSize, lossCount, spinData[row+i]);
    }

    printf("Purse: %d\n", purse);
}

// http://www.maths.surrey.ac.uk/hosted-sites/R.Knott/Fibonacci/fibFormula.html
__device__ int calculateFibonacciNumber(int n)
{
    return round((pow(PHI, n) - pow(-PHI, -n)) / sqrtf(5));
}

__global__ void fibonacci(float winProbability, hiprandState_t* states, float* spinData, int spinsPerRun, int bettingFactor = 2)
{
    genRandoms(states, spinData, spinsPerRun);

    int tid = (blockDim.x * blockIdx.x) + threadIdx.x;
    int row = tid * spinsPerRun;
    int purse = 0;
    int betSize = 1;
    int lossCount = 0;
    int winLossFactor[] = {1, -1};
    int totalLosses = 0;

    printf("Win probability: %f\n\n", winProbability);
    for (int i = 0; i < spinsPerRun; i++)
    {
        printf("!!Begin!! TID: %d -- Run: %d -- Purse: %d -- Bet: %d -- Losses: %d -- Spin: %f\n", tid, i, purse, betSize, lossCount, spinData[row+i]);
        int lostSpin = (spinData[row + i] >= winProbability);
        purse += winLossFactor[lostSpin] * betSize;

        lossCount = lossCount * lostSpin + lostSpin;
        totalLosses += (lossCount > 0);
        betSize = bettingFactor * calculateFibonacciNumber(lossCount + 1);
        printf("!!END  !! TID: %d -- Run: %d -- Purse: %d -- Bet: %d -- Losses: %d -- Spin: %f\n\n", tid, i, purse, betSize, lossCount, spinData[row+i]);
    }

    printf("Purse: %d\n", purse);
    printf("TotalLosses %d\n", totalLosses);
}

hiprandState_t* initializeRandom(int numRuns)
{
  hiprandState_t* states;
  hipMalloc((void**) &states, numRuns * sizeof(hiprandState_t));

  //init<<<numRuns / BLOCK_SIZE, BLOCK_SIZE>>>(time(0), states);
  initRandom<<<1, numRuns>>>(time(0), states);

  return states;
}

void playRoulette(int numRuns, int spinsPerRun, float winProbability, BettingStrategy strategy, int bettingFactor = 2)
{
    // Get the average of a set of random numbers
    auto start = std::chrono::high_resolution_clock::now();
    hiprandState_t* states = initializeRandom(numRuns);

    float * spinData;
    hipMalloc((void**) &spinData, numRuns * spinsPerRun * sizeof(float));
    if (strategy == MARTINGALE)
    {
        martingale<<<1, numRuns>>>(winProbability, states, spinData, spinsPerRun, bettingFactor);
    }
    else if (strategy == DALEMBERT)
    {
        dalembert<<<1, numRuns>>>(winProbability, states, spinData, spinsPerRun, bettingFactor);
    }
    else if (strategy == FIBONACCI)
    {
        fibonacci<<<1, numRuns>>>(winProbability, states, spinData, spinsPerRun, bettingFactor);
    }
    hipDeviceSynchronize();

    hipFree(states);
    auto stop = std::chrono::high_resolution_clock::now();

    auto runTime = std::chrono::duration_cast<std::chrono::microseconds>(stop - start).count();
    printf("It took %d us \n", runTime);
}

// Main function
int main(int argc, char* argv[])
{
    ProgramOptions options = parseOptions(argc, argv);
    playRoulette(options.numRuns, options.spinsPerRun, options.winProbability, options.bettingStrategy, options.bettingFactor);
}
