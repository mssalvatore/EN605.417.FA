#include <chrono>
#include <cmath>
#include <cstdlib>
#include <iostream>
#include <fstream>
#include <stdlib.h>
#include <stdio.h>
#include <time.h>

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include "options.h"
#include "util.cu"
#include "commonBetting.cu"
#include "dalembert.cu"
#include "fibonacci.cu"
#include "martingale.cu"
#include "analytics.h"

#define MAX_THREADS_PER_BLOCK 1024
#define BLOCK_SIZE MAX_THREADS_PER_BLOCK
#define WARP_SIZE 32
#define NUM_ROULETTE_SLOTS 37

hiprandState_t* initializeRandom(int numRuns)
{
  hiprandState_t* states;
  hipMalloc((void**) &states, numRuns * sizeof(hiprandState_t));

  //init<<<numRuns / BLOCK_SIZE, BLOCK_SIZE>>>(time(0), states);
  initRandom<<<1, numRuns>>>(time(0), states);

  return states;
}

template <class T>
void printArray(T * data, size_t size)
{
    for (int i = 0; i < size; i++)
    {
        std::cout<<data[i] << " ";
    }
    std::cout<<std::endl;
}

void playRoulette(float * spinData, int numBlocks, int numThreads, int spinsPerRun, float winProbability, BettingStrategy strategy, int bettingFactor = 2)
{
    int * gpuPurse;
    int * gpuMaxPurse;
    int * gpuMinPurse;
    int64_t * gpuIntegral;
    hipMalloc((void**)&gpuPurse, numBlocks * numThreads * sizeof(int));
    hipMalloc((void**)&gpuMaxPurse, numBlocks * numThreads * sizeof(int));
    hipMalloc((void**)&gpuMinPurse, numBlocks * numThreads * sizeof(int));
    hipMalloc((void**)&gpuIntegral, numBlocks * numThreads * sizeof(int64_t));

    auto start = std::chrono::high_resolution_clock::now();
    if (strategy == MARTINGALE)
    {
        martingale<<<numBlocks, numThreads>>>(gpuPurse, gpuMaxPurse, gpuMinPurse, gpuIntegral, winProbability, spinData, spinsPerRun, bettingFactor);
    }
    else if (strategy == DALEMBERT)
    {
        dalembert<<<numBlocks, numThreads>>>(gpuPurse, gpuMaxPurse, gpuMinPurse, gpuIntegral, winProbability, spinData, spinsPerRun, bettingFactor);
    }
    else if (strategy == FIBONACCI)
    {
        fibonacci<<<numBlocks, numThreads>>>(gpuPurse, gpuMaxPurse, gpuMinPurse, gpuIntegral, winProbability, spinData, spinsPerRun, bettingFactor);
    }
    hipDeviceSynchronize();

    auto stop = std::chrono::high_resolution_clock::now();

    auto runTime = std::chrono::duration_cast<std::chrono::microseconds>(stop - start).count();
    printf("It took %d us \n", runTime);

    int * purse = (int*) malloc(numBlocks * numThreads * sizeof(int));
    int * maxPurse = (int*) malloc(numBlocks * numThreads * sizeof(int));
    int * minPurse = (int*) malloc(numBlocks * numThreads * sizeof(int));
    int64_t * integral = (int64_t*) malloc(numBlocks * numThreads * sizeof(int64_t));

    hipMemcpy(purse, gpuPurse, numBlocks * numThreads * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(maxPurse, gpuMaxPurse, numBlocks * numThreads * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(minPurse, gpuMinPurse, numBlocks * numThreads * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(integral, gpuIntegral, numBlocks * numThreads * sizeof(int64_t), hipMemcpyDeviceToHost);

    hipFree(gpuPurse);
    hipFree(gpuMaxPurse);
    hipFree(gpuMinPurse);
    hipFree(gpuIntegral);

    runAnalytics(purse, maxPurse, minPurse, integral, numBlocks * numThreads);
}

float * prepareRandomData(int numBlocks, int numThreads, int spinsPerRun)
{
    hiprandState_t* states = initializeRandom(numBlocks * numThreads);

    float * spinData;
    hipMalloc((void**) &spinData, numBlocks * numThreads * spinsPerRun * sizeof(float));

    genRandoms<<<numBlocks, numThreads>>>(states, spinData, spinsPerRun);

    hipFree(states);
    return spinData;
}

float * prepareRealData(char * fileName)
{
    std::ifstream file(fileName);
    if (!file) {
        std::cout << "Error opening input file " << fileName << std::endl;
        exit(1);
    }

    float *gpuRealSpins;
    float *realSpins;
    int numBlocks;
    int numThreads;
    std::vector<int> realSpinsFromFile = readLineFromFile(&file);
    size_t numRealSpins = realSpinsFromFile.size();

    getDimensions(&numBlocks, &numThreads, MAX_THREADS_PER_BLOCK, numRealSpins);

    hipHostMalloc((void**)&realSpins, realSpinsFromFile.size() * sizeof(float));
    std::copy(realSpinsFromFile.begin(), realSpinsFromFile.end(), realSpins);

    hipMemcpyToSymbol(HIP_SYMBOL(cudaColorTranslation), &hostColorTranslation, NUM_ROULETTE_SLOTS * sizeof(float));
    hipMalloc((void **)&gpuRealSpins, numRealSpins * sizeof(float));
    hipMemcpy(gpuRealSpins, realSpins, numRealSpins * sizeof(float), hipMemcpyHostToDevice);

    translateSpinsToColors<<<numBlocks,numThreads>>>(gpuRealSpins);

    return gpuRealSpins;
}

// Main function
int main(int argc, char* argv[])
{
    ProgramOptions options = parseOptions(argc, argv);
    /*
    std::cout<<"numBlocks " << options.numBlocks <<std::endl;
    std::cout<<"numThreads " << options.numThreads <<std::endl;
    std::cout<<"spinsPerRun " << options.spinsPerRun <<std::endl;
    std::cout<<"winProbability " << options.winProbability <<std::endl;
    std::cout<<"bettingFactor " << options.bettingFactor <<std::endl;
    std::cout<<"bettingStrategy " << options.bettingStrategy <<std::endl;
    std::cout<<"fileName " << options.fileName <<std::endl;
    */
    float * spinData;
    if (options.fileName)
    {
        spinData = prepareRealData(options.fileName);
    }
    else {
        spinData = prepareRandomData(options.numBlocks, options.numThreads, options.spinsPerRun);
    }
    playRoulette(spinData, options.numBlocks, options.numThreads, options.spinsPerRun, options.winProbability, options.bettingStrategy, options.bettingFactor);

    hipFree(spinData);
    exit(0);
}
