#include <chrono>
#include <cmath>
#include <cstdlib>
#include <iostream>
#include <stdlib.h>
#include <stdio.h>
#include <time.h>

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include "options.h"
#include "util.cu"
#include "commonBetting.cu"
#include "dalembert.cu"
#include "fibonacci.cu"
#include "martingale.cu"

#define MAX_THREADS_PER_BLOCK 1024
#define BLOCK_SIZE MAX_THREADS_PER_BLOCK
#define WARP_SIZE 32

hiprandState_t* initializeRandom(int numRuns)
{
  hiprandState_t* states;
  hipMalloc((void**) &states, numRuns * sizeof(hiprandState_t));

  //init<<<numRuns / BLOCK_SIZE, BLOCK_SIZE>>>(time(0), states);
  initRandom<<<1, numRuns>>>(time(0), states);

  return states;
}

void playRoulette(int numRuns, int spinsPerRun, float winProbability, BettingStrategy strategy, int bettingFactor = 2)
{
    // Get the average of a set of random numbers
    auto start = std::chrono::high_resolution_clock::now();
    hiprandState_t* states = initializeRandom(numRuns);

    float * spinData;
    hipMalloc((void**) &spinData, numRuns * spinsPerRun * sizeof(float));
    if (strategy == MARTINGALE)
    {
        martingale<<<1, numRuns>>>(winProbability, states, spinData, spinsPerRun, bettingFactor);
    }
    else if (strategy == DALEMBERT)
    {
        dalembert<<<1, numRuns>>>(winProbability, states, spinData, spinsPerRun, bettingFactor);
    }
    else if (strategy == FIBONACCI)
    {
        fibonacci<<<1, numRuns>>>(winProbability, states, spinData, spinsPerRun, bettingFactor);
    }
    hipDeviceSynchronize();

    hipFree(states);
    auto stop = std::chrono::high_resolution_clock::now();

    auto runTime = std::chrono::duration_cast<std::chrono::microseconds>(stop - start).count();
    printf("It took %d us \n", runTime);
}

// Main function
int main(int argc, char* argv[])
{
    ProgramOptions options = parseOptions(argc, argv);
    playRoulette(options.numRuns, options.spinsPerRun, options.winProbability, options.bettingStrategy, options.bettingFactor);
}
