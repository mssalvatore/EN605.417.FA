typedef int (*betsize_calculator_function)(int bettingFactor, int lossCount);
typedef int (*loss_calculator_function)(int currentLossCount, int spinResult, int winLossFactor[]);

__device__ void executeBettingStrategy(loss_calculator_function calcLossCount, betsize_calculator_function calcBetSize, float winProbability, hiprandState_t* states, float* spinData, int spinsPerRun, int bettingFactor = 2, int startingBet = 1)
{
    genRandoms(states, spinData, spinsPerRun);

    int tid = (blockDim.x * blockIdx.x) + threadIdx.x;
    int row = tid * spinsPerRun;
    int purse = 0;
    int betSize = startingBet;
    int lossCount = 0;
    int winLossFactor[] = {1, -1};
    int totalLosses = 0;

    printf("Win probability: %f\n\n", winProbability);
    printf("bettingFactor : %d\n\n", bettingFactor);
    for (int i = 0; i < spinsPerRun; i++)
    {
        printf("!!Begin!! TID: %d -- Run: %d -- Purse: %d -- Bet: %d -- Losses: %d -- Spin: %f\n", tid, i, purse, betSize, lossCount, spinData[row+i]);
        int lostSpin = (spinData[row + i] >= winProbability);
        purse += winLossFactor[lostSpin] * betSize;

        lossCount = calcLossCount(lossCount, lostSpin, winLossFactor);
        totalLosses += lostSpin;
        betSize = calcBetSize(bettingFactor, lossCount);
        printf("!!END  !! TID: %d -- Run: %d -- Purse: %d -- Bet: %d -- Losses: %d -- Spin: %f\n\n", tid, i, purse, betSize, lossCount, spinData[row+i]);
    }

    printf("Purse: %d\n", purse);
    printf("TotalLosses %d\n", totalLosses);
}

__device__ int lossCountResetOnWin(int currentLossCount, int spinResult, int * /*[] winLossFactor */)
{
    return currentLossCount * spinResult + spinResult;
}

