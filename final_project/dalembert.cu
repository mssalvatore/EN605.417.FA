#include "hip/hip_runtime.h"
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

__device__ int calculateDalembertLossCount(int currentLossCount, int spinResult, int winLossFactor[])
{
        int lossCount = (currentLossCount + winLossFactor[!spinResult]);
        return lossCount * (lossCount > 0);
}

__device__ int calculateDalembertBetSize(int bettingFactor, int lossCount)
{
        return bettingFactor + (bettingFactor * lossCount);
}

__global__ void dalembert(int * outPurse, int * outMaxPurse, int * outMinPurse, int64_t * outIntegral, float winProbability, float* spinData, int spinsPerRun, int bettingFactor = 1)
{
    executeBettingStrategy(outPurse, outMaxPurse, outMinPurse, outIntegral, &calculateDalembertLossCount, &calculateDalembertBetSize, winProbability, spinData, spinsPerRun, bettingFactor);
}

