
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdint.h>
#include <iostream>
#include <chrono>

const uint32_t MAX_DATA_SET_SIZE_PER_BLOCK = 1024;
const uint32_t DEFAULT_DATA_SIZE = 8192;

typedef uint32_t (*operation_function)(uint32_t operand1, uint32_t operand2);

// Reduction adapted from http://developer.download.nvidia.com/compute/cuda/1.1-Beta/x86_website/projects/reduction/doc/reduction.pdf
__device__ void reduce(uint32_t *dataSet, uint32_t* sharedData, uint32_t *result, operation_function op)
{
    unsigned int threadId = threadIdx.x;
    unsigned int globalThreadId = blockIdx.x*(blockDim.x*2) + threadIdx.x;
    sharedData[threadId] = op(dataSet[globalThreadId], dataSet[globalThreadId + blockDim.x]);
    __syncthreads();

    for (unsigned int i=blockDim.x/2; i>0; i >>= 1)
    {
        if (threadId < i)
        {
            sharedData[threadId] = op(sharedData[threadId], sharedData[threadId + i]);
        }
        __syncthreads();
    }

    if (threadId == 0)
    {
        result[blockIdx.x] = sharedData[0];
    }
}

__device__ uint32_t add(uint32_t operand1, uint32_t operand2)
{
    return operand1 + operand2;
}

__device__ uint32_t maximum(uint32_t operand1, uint32_t operand2)
{
    uint32_t findMax[2];
    findMax[0] = operand1;
    findMax[1] = operand2;

    return findMax[operand1 < operand2];
}

__device__ uint32_t minimum(uint32_t operand1, uint32_t operand2)
{
    uint32_t findMax[2];
    findMax[0] = operand1;
    findMax[1] = operand2;

    return findMax[operand1 > operand2];
}

__global__ void sum(uint32_t* dataSet, uint32_t *result)
{
    extern __shared__ uint32_t sharedData[];
    reduce(dataSet, sharedData, result, &add);
}

__global__ void getMax(uint32_t* dataSet, uint32_t *result)
{
    extern __shared__ uint32_t sharedData[];
    reduce(dataSet, sharedData, result, &maximum);
}

__global__ void getMin(uint32_t* dataSet, uint32_t *result)
{
    extern __shared__ uint32_t sharedData[];
    reduce(dataSet, sharedData, result, &minimum);
}

void getMaxMinAvg(uint32_t dataSize, uint32_t numBlocks, uint32_t * data, uint32_t *max, uint32_t *min, double *avg)
{
    int32_t numThreads = dataSize / (numBlocks * 2);
    uint32_t *cpuMaxResult = (uint32_t *)malloc(numBlocks * sizeof(uint32_t));
    uint32_t *cpuMinResult = (uint32_t *)malloc(numBlocks * sizeof(uint32_t));
    uint32_t *cpuSumResult = (uint32_t *)malloc(numBlocks * sizeof(uint32_t));

    uint32_t *gpuMaxResult;
    hipMalloc((void **)&gpuMaxResult, numBlocks * sizeof(uint32_t));
    uint32_t *gpuMinResult;
    hipMalloc((void **)&gpuMinResult, numBlocks * sizeof(uint32_t));
    uint32_t *gpuSumResult;
    hipMalloc((void **)&gpuSumResult, numBlocks * sizeof(uint32_t));

    hipEvent_t gotMax, gotMin, gotSum;
    hipEventCreate(&gotMax);
    hipEventCreate(&gotMin);
    hipEventCreate(&gotSum);

    hipStream_t maxStream, minStream, sumStream;
    hipStreamCreate(&maxStream);
    hipStreamCreate(&minStream);
    hipStreamCreate(&sumStream);

    auto start = std::chrono::high_resolution_clock::now();
    getMax<<<numBlocks, numThreads, numThreads * sizeof(uint32_t), maxStream>>>(data, gpuMaxResult);
    hipMemcpyAsync(cpuMaxResult, gpuMaxResult, numBlocks * sizeof(uint32_t), hipMemcpyDeviceToHost, maxStream);
    hipEventRecord(gotMax, maxStream);

    getMin<<<numBlocks, numThreads, numThreads * sizeof(uint32_t), minStream>>>(data, gpuMinResult);
    hipMemcpyAsync(cpuMinResult, gpuMinResult, numBlocks * sizeof(uint32_t), hipMemcpyDeviceToHost, minStream);
    hipEventRecord(gotMin, minStream);

    sum<<<numBlocks, numThreads, numThreads * sizeof(uint32_t), sumStream>>>(data, gpuSumResult);
    hipMemcpyAsync(cpuSumResult, gpuSumResult, numBlocks * sizeof(uint32_t), hipMemcpyDeviceToHost, sumStream);
    hipEventRecord(gotSum, sumStream);

    bool maxFinished = false;
    bool minFinished = false;
    bool sumFinished = false;

    while (!maxFinished && !minFinished && !sumFinished)
    {
        if (!maxFinished && (hipEventQuery(gotMax) == hipSuccess))
        {
            auto now = std::chrono::high_resolution_clock::now();
            printf("Get Maximum finished after %dus\n", std::chrono::duration_cast<std::chrono::microseconds>(now - start).count());
            maxFinished = true;
        }
        if (!minFinished && (hipEventQuery(gotMin) == hipSuccess))
        {
            auto now = std::chrono::high_resolution_clock::now();
            printf("Get Minimum finished after %dus\n", std::chrono::duration_cast<std::chrono::microseconds>(now - start).count());
            minFinished = true;
        }
        if (!sumFinished && (hipEventQuery(gotSum) == hipSuccess))
        {
            auto now = std::chrono::high_resolution_clock::now();
            printf("Get Sum finished after %dus\n", std::chrono::duration_cast<std::chrono::microseconds>(now - start).count());
            sumFinished = true;
        }
    }
    hipDeviceSynchronize();

    *max = 0;
    *min = UINT_MAX;
    double sum = 0;

    for (size_t i = 0; i < numBlocks; i++)
    {
        sum += cpuSumResult[i];

        if (cpuMaxResult[i] > *max) {
            *max = cpuMaxResult[i];
        }
        if (cpuMinResult[i] < *min) {
            *min = cpuMinResult[i];
        }
    }

    *avg = sum / dataSize;

    hipEventDestroy(gotMax);
    hipEventDestroy(gotMin);
    hipEventDestroy(gotSum);

    hipStreamDestroy(maxStream);
    hipStreamDestroy(minStream);
    hipStreamDestroy(sumStream);

    hipFree(gpuMaxResult);
    hipFree(gpuMinResult);
    hipFree(gpuSumResult);
}

int main(int argc, char* argv[])
{
    uint32_t dataSize = DEFAULT_DATA_SIZE;
    uint32_t numBlocks = 8;
    uint32_t numRuns = 2;
    if (argc > 1) {
        numBlocks = atoi(argv[1]);

        if ((numBlocks % 2) != 0) {
            printf("Must enter a multiple of 2\n");
            return 1;
        }
        dataSize = numBlocks * MAX_DATA_SET_SIZE_PER_BLOCK ;
    }

    if (argc > 2) {
        numRuns = atoi(argv[2]);
    }

    uint32_t * data;
    hipHostMalloc((void**)&data, dataSize * sizeof(uint32_t), hipHostMallocDefault);

    uint32_t *gpuData;
    hipMalloc((void **)&gpuData, dataSize * sizeof(uint32_t));

    srand((unsigned)time(NULL)); 
    for (uint32_t x = 0; x < numRuns; x++)
    {
        printf("Run %d\n--------------------------------------------------\n", x);

        uint32_t range = dataSize * 4;
        for(size_t i = 0; i < dataSize; i++){ 
            data[i] = rand() % range + 1;
        }

        hipMemcpy(gpuData, data, dataSize * sizeof(uint32_t), hipMemcpyHostToDevice);

        double avg;
        uint32_t max;
        uint32_t min;
        getMaxMinAvg(dataSize, numBlocks, gpuData, &max, &min, &avg);

        printf("\n\n");
        printf("Average is %f\n", avg);
        printf ("Max is %d\n", max);
        printf ("Min is %d\n", min);
        printf("\n\n\n");
    }

    hipHostFree(data);
    hipFree(gpuData);
}
