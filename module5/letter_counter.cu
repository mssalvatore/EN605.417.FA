// DESCRIPTION HERE

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdint.h>
#include <iostream>
#include <chrono>

#define NUM_THREADS 448
#define NUM_CHARS 26
#define SHARED_MEM_SIZE NUM_THREADS * NUM_CHARS

// Calculate how much padding is needed to make the file evenly divided into 448 chunks
size_t calculatePadding(size_t fileSize)
{
	size_t paddingBytes = NUM_THREADS - (fileSize % NUM_THREADS);
	return paddingBytes;
}

// Pad the data so it is evenly divided into 448 chunks
void padData(uint8_t * buf, size_t bytesRead, size_t numPadBytes)
{
    for (size_t i = 0; i < numPadBytes; i++)
    {
        buf[bytesRead + i] = 'a';
    }
}

// Read a file into a byte array
uint8_t * readFile(const char * filename, size_t * outBytesRead, size_t * paddingBytes)
{
	FILE *handle = fopen(filename, "rb");
	fseek(handle, 0, SEEK_END);
	*outBytesRead = ftell(handle);
	*paddingBytes = calculatePadding(*outBytesRead);
	rewind(handle);

	uint8_t * buf = (uint8_t *) malloc((*outBytesRead + *paddingBytes)*sizeof(uint8_t));
	fread(buf, *outBytesRead, 1, handle);
	fclose(handle);

    padData(buf, *outBytesRead, *paddingBytes);

	return buf;
}

// Shift all ascii letters so that 'a' is index 0, 'b' is index 1, etc.
__device__ __constant__ int shiftAmount;
__global__ void shiftLetters(uint8_t *data)
{
    uint32_t threadId = blockIdx.x * blockDim.x + threadIdx.x;
    data[threadId] = data[threadId] - shiftAmount;
}

// Zero out the letter counts
__device__ void zeroLetterCounts(uint32_t * letterCounts)
{
    for (size_t i = 0; i < NUM_CHARS; i++) {
        letterCounts[(threadIdx.x * NUM_CHARS) + i] = 0;
    }
}

// Count the occurence of each letter in *data
__device__ void countLetters(uint8_t *data, uint32_t *letterCounts, uint32_t *threadLetterCounts, size_t chunkSize)
{

    zeroLetterCounts(threadLetterCounts);
    __syncthreads();

    // Tally letters for each thread
    for (size_t i = 0; i < chunkSize; i++)
    {
        threadLetterCounts[(threadIdx.x * NUM_CHARS) + data[(threadIdx.x * chunkSize) + i]]++;
    }

    __syncthreads();

    // Total local thread tallys
    if (threadIdx.x < NUM_CHARS)
    {
        for (size_t i = 0; i < NUM_THREADS; i++)
        {
            letterCounts[threadIdx.x] += threadLetterCounts[threadIdx.x + (i * NUM_CHARS)]; 
        }
    }
}

// Count the occurence of each letter in *data using shared memory
__global__ void countLettersShared(uint8_t *data, uint32_t *letterCounts, size_t chunkSize)
{
    __shared__ uint32_t sharedLetterCounts[SHARED_MEM_SIZE];
    countLetters(data, letterCounts, sharedLetterCounts, chunkSize);
}

// Count the occurence of each letter in *data using global memory
__global__ void countLettersGlobal(uint8_t *data, uint32_t *letterCounts, uint32_t * threadLetterCounts, size_t chunkSize)
{
    countLetters(data, letterCounts, threadLetterCounts, chunkSize);
}

// Remove any padding so that letter counts are accurage
void unpadResult(uint32_t * letterCounts, size_t paddingBytes)
{
    letterCounts[0] -= paddingBytes;
}

// Count the occurence of each letter in *data using shared memory
uint64_t countWithGPUShared(uint8_t * data, size_t dataSize, uint32_t * letterCounts, size_t textChunkSize)
{
    // Declare cuda memory
	uint8_t *gpuData;
    uint32_t *gpuLetterCounts;
	hipMalloc((void **)&gpuData, dataSize);
	hipMemcpy(gpuData, data, dataSize, hipMemcpyHostToDevice);

	hipMalloc((void **)&gpuLetterCounts, NUM_CHARS * sizeof(uint32_t));
	hipMemcpy(gpuLetterCounts, letterCounts, NUM_CHARS * sizeof(uint32_t), hipMemcpyHostToDevice);

    // Run Kernel
    auto start = std::chrono::high_resolution_clock::now();
    shiftLetters<<<textChunkSize, NUM_THREADS>>>(gpuData);
    countLettersShared<<<1, NUM_THREADS>>>(gpuData, gpuLetterCounts, textChunkSize);
    auto stop = std::chrono::high_resolution_clock::now();

	hipMemcpy(letterCounts, gpuLetterCounts, NUM_CHARS * sizeof(uint32_t), hipMemcpyDeviceToHost);

	// Free the arrays on the GPU as now we're done with them
	hipFree(gpuData);
	hipFree(gpuLetterCounts);

    return std::chrono::duration_cast<std::chrono::nanoseconds>(stop - start).count();
}

// Count the occurence of each letter in *data using global memory
uint64_t countWithGPUGlobal(uint8_t * data, size_t dataSize, uint32_t * letterCounts, size_t textChunkSize)
{
    // Declare cuda memory
	uint8_t *gpuData;
    uint32_t *gpuLetterCounts;
    uint32_t *threadLetterCounts;
	hipMalloc((void **)&gpuData, dataSize);
	hipMemcpy(gpuData, data, dataSize, hipMemcpyHostToDevice);

	hipMalloc((void **)&gpuLetterCounts, NUM_CHARS * sizeof(uint32_t));
	hipMemcpy(gpuLetterCounts, letterCounts, NUM_CHARS * sizeof(uint32_t), hipMemcpyHostToDevice);

	hipMalloc((void **)&threadLetterCounts, SHARED_MEM_SIZE);

    // Run Kernel
    auto start = std::chrono::high_resolution_clock::now();
    shiftLetters<<<textChunkSize, NUM_THREADS>>>(gpuData);
    countLettersGlobal<<<1, NUM_THREADS>>>(gpuData, gpuLetterCounts, threadLetterCounts, textChunkSize);
    auto stop = std::chrono::high_resolution_clock::now();

	hipMemcpy(letterCounts, gpuLetterCounts, NUM_CHARS * sizeof(uint32_t), hipMemcpyDeviceToHost);

	/* Free the arrays on the GPU as now we're done with them */
	hipFree(gpuData);
	hipFree(gpuLetterCounts);

    return std::chrono::duration_cast<std::chrono::nanoseconds>(stop - start).count();
}

// Use the CPU to count the occurrences of each letter in *data
uint64_t countWithCPU(uint8_t * data, size_t dataSize, uint32_t * letterCounts, int ascii_a)
{
    auto start = std::chrono::high_resolution_clock::now();
    for (size_t i = 0; i < dataSize; i++)
    {
        letterCounts[data[i] - ascii_a]++;
    }
    auto stop = std::chrono::high_resolution_clock::now();

    return std::chrono::duration_cast<std::chrono::nanoseconds>(stop - start).count();
}

// Display letter counts
void displayResults(uint32_t * letterCounts)
{
    printf("\n\n");
    for (size_t i = 0; i < NUM_CHARS; i++)
    {
        printf("Found %d %c's\n", letterCounts[i], i + 0x61);
    }

    printf("\n\n");
}

// Display and analyze the run times (shared vs. global vs. CPU)
void displayTimingResults(uint64_t gpuSharedDuration, uint64_t gpuGlobalDuration, uint64_t cpuDuration)
{
    printf("Took %dns to run processing on GPU with shared memory\n", gpuSharedDuration);
    printf("Took %dns to run processing on GPU with global memory\n", gpuGlobalDuration);
    printf("Took %dns to run on CPU\n", cpuDuration);

    printf("\n");
    printf("Shared Memory runs %fx faster than global memory\n", ((double)gpuGlobalDuration) / gpuSharedDuration);
    printf("Shared Memory on GPU runs %fx faster than the CPU\n", ((double)cpuDuration) / gpuSharedDuration);
    printf("\n");
}

int main(int argc, char* argv[])
{
    // Read command line args
	std::string fileName = "all_letter.shakespeare.txt";
	if (argc > 1) {
		fileName = argv[1];
	}

    // Copy from host to constant memory
    const int ascii_a = 0x61;
    hipMemcpyToSymbol(HIP_SYMBOL(shiftAmount), &ascii_a, sizeof(uint8_t));

    // Declare some variables
    uint32_t letterCounts[NUM_CHARS];
	size_t bytesRead;
	size_t paddingBytes;
    
    // Read file
	uint8_t * data = readFile(fileName.c_str(), &bytesRead, &paddingBytes);

    // Calculate run-time parameters
	size_t dataSize = bytesRead + paddingBytes;
    size_t textChunkSize = dataSize / NUM_THREADS;
	printf("Bytes read: %d\n", bytesRead);
	printf("Padding bytes: %d\n", paddingBytes);

    uint8_t *pinnedData;
    hipHostMalloc((void**)&pinnedData, dataSize, hipHostMallocDefault);
    memcpy(pinnedData, data, dataSize);

    // Run letter counter on the CPU
    memset(letterCounts, 0, NUM_CHARS * sizeof(uint32_t));
    uint64_t cpuDuration = countWithCPU(pinnedData, dataSize, letterCounts, ascii_a);

    // Run letter counter on the GPU with global memory
    memset(letterCounts, 0, NUM_CHARS * sizeof(uint32_t));
    uint64_t gpuGlobalDuration = countWithGPUGlobal(pinnedData, dataSize, letterCounts, textChunkSize);

    // Run letter counter on the GPU with shared memory
    memset(letterCounts, 0, NUM_CHARS * sizeof(uint32_t));
    uint64_t gpuSharedDuration = countWithGPUShared(pinnedData, dataSize, letterCounts, textChunkSize);
    unpadResult(letterCounts, paddingBytes);

    // Display letter counts and timing
    displayResults(letterCounts);
    displayTimingResults(gpuSharedDuration, gpuGlobalDuration, cpuDuration);

	return EXIT_SUCCESS;
}
